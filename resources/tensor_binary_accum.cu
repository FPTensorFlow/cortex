#include "hip/hip_runtime.h"
#include "index_system.h"
#include "datatypes.h"
#include "operations.h"
#include "cuda_convert.h"

using namespace think;
using namespace tensor::index_system;
using namespace tensor::operations;



template<typename dtype>
__device__
void binary_accum(dtype* dest, const general_index_system& dest_sys, dtype dest_alpha,
		  const dtype* rhs, const general_index_system& rhs_sys, dtype rhs_alpha,
		  const general_operation& operation,
		  int n_elems)
{
  typedef Converter<dtype> TConverterType;
  typedef typename TConverterType::rettype TIntType;
  int elem_idx = blockDim.x * blockIdx.x + threadIdx.x;
  if ( elem_idx < n_elems ) {
    int dest_idx = dest_sys(elem_idx);
    dtype rhs_val = rhs_alpha * rhs[rhs_sys(elem_idx)];
    dtype* dest_ptr = dest + dest_idx;
    TIntType* int_addr = TConverterType::from(dest_ptr);
    volatile dtype* write_ptr(dest_ptr);
    TIntType old, assumed;
    do {
      dtype dest_val = *write_ptr;
      assumed = TConverterType::from(dest_val);
      dtype new_val = operation(dest_val * dest_alpha, rhs_val);
      old = atomicCAS(int_addr, assumed, TConverterType::from(new_val));
    } while (assumed != old);
  }
}


#define DATATYPE_ITERATOR(dtype,export_sym)				\
  extern "C"								\
  __global__								\
  void tensor_binary_accum##export_sym(					\
    datatype::dtype* dest, EXPLODE_IDX_SYSTEM(dest), datatype::dtype dest_alpha, \
    const datatype::dtype* rhs, EXPLODE_IDX_SYSTEM(rhs), datatype::dtype rhs_alpha, \
    EXPLODE_OP_SYSTEM_REV(bin_op), int n_elems) {			\
    binary_accum( dest, ENCAPSULATE_IDX_SYSTEM(dest), dest_alpha,	\
		  rhs, ENCAPSULATE_IDX_SYSTEM(rhs), rhs_alpha,		\
		  ENCAPSULATE_OP_SYSTEM_REV(bin_op),			\
		  n_elems );						\
  }

ITERATE_DATATYPES_EXPORT_CAS;
#undef DATATYPE_ITERATOR
